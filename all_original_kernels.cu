#include "hip/hip_runtime.h"
#include <stdio.h>

#include <kernels.cuh>

#define XSIZE 7
#define YSIZE 128
#define ZSIZE 48

// __restrict__ tells the compiler there is no memory overlap

__device__ float fftfactor = 1.0/32.0 * 1.0/32.0;

__global__ void rearrange(hipTextureObject_t texObj, hipfftComplex * __restrict__ out)
{
    // this is currently the ugliest solution I can think of
    // xidx is the channel number
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * 128;
    int2 word;
    //if ((xidx == 0) && (yidx == 0)) printf("In the rearrange kernel\n");
    for (int sample = 0; sample < YSIZE; sample++) {
         word = tex2D<int2>(texObj, xidx, yidx + sample);
         printf("%i ", sample);
         out[xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
         out[xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
    }
}

__global__ void rearrange2(hipTextureObject_t texObj, hipfftComplex * __restrict__ out, unsigned int acc)
{

    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * 128;
    int chanidx = threadIdx.x + blockIdx.y * 7;
    int skip;
    int2 word;

    for (int ac = 0; ac < acc; ac++) {
        skip = 336 * 128 * 2 * ac;
        for (int sample = 0; sample < YSIZE; sample++) {
            word = tex2D<int2>(texObj, xidx, yidx + ac * 48 * 128 + sample);
            out[skip + chanidx * YSIZE * 2 + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
            out[skip + chanidx * YSIZE * 2 + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
            out[skip + chanidx * YSIZE * 2 + YSIZE + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
            out[skip + chanidx * YSIZE * 2 + YSIZE + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
        }
    }
}


__global__ void addtime(float *in, float *out, unsigned int jumpin, unsigned int jumpout, unsigned int factort)
{

    // index will tell which 1MHz channel we are taking care or
    // use 1 thread per 1MHz channel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx == 0) printf("In the time kernel\n");

    for(int ch = 0; ch < 27; ch++) {
	// have to restart to 0, otherwise will add to values from previous execution
        out[idx * 27 + ch] = (float)0.0;
        out[idx * 27 + ch + jumpout] = (float)0.0;
        out[idx * 27 + ch + 2 * jumpout] = (float)0.0;
        out[idx * 27 + ch + 3 * jumpout] = (float)0.0;

        for (int t = 0; t < factort; t++) {
            out[idx * 27 + ch] += in[idx * 128 + ch + t * 32];
            //printf("S1 time sum %f\n", out[idx * 27 + ch]);
            out[idx * 27 + ch + jumpout] += in[idx * 128 + ch + t * 32 + jumpin];
            out[idx * 27 + ch + 2 * jumpout] += in[idx * 128 + ch + t * 32 + 2 * jumpin];
            out[idx * 27 + ch + 3 * jumpout] += in[idx * 128 + ch + t * 32 + 3 * jumpin];
        }
    }
}

/*__global__ void addtime(float* __restrict__ int, float* __restrict__ out, unsigned int jumpin, unsigned int jumpout, unsigned int factort)
{


} */

__global__ void addchannel(float* __restrict__ in, float* __restrict__ out, unsigned int jumpin, unsigned int jumpout, unsigned int factorc) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx == 0) printf("In the channel kernel\n");

    out[idx] = (float)0.0;
    out[idx + jumpout] = (float)0.0;
    out[idx + 2 * jumpout] = (float)0.0;
    out[idx + 3 * jumpout] = (float)0.0;

    for (int ch = 0; ch < factorc; ch++) {
        out[idx] += in[idx * factorc + ch];
        out[idx + jumpout] += in[idx * factorc + ch + jumpin];
        out[idx + 2 * jumpout] += in[idx * factorc + ch + 2 * jumpin];
        out[idx + 3 * jumpout] += in[idx * factorc + ch + 3 * jumpin];
    }

    //printf("S1 freq sum %f\n", out[idx]);
}

__global__ void addchannel2(float* __restrict__ in, float** __restrict__ out, short nchans, size_t gulp, size_t totsize,  short gulpno, unsigned int jumpin, unsigned int factorc, unsigned int framet, unsigned int acc) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int extra = totsize - gulpno * gulp;
    // thats the starting save position for the chunk of length acc time samples
    int saveidx;

    int inskip;

    for (int ac = 0; ac < acc; ac++) {
        saveidx = (framet % (gulpno * gulp)) * nchans + idx;
        inskip = ac * 27 * 336;

        out[0][saveidx] = (float)0.0;
        out[1][saveidx] = (float)0.0;
        out[2][saveidx] = (float)0.0;
        out[3][saveidx] = (float)0.0;

        if ((framet % (gulpno * gulp)) >= extra) {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][saveidx] += in[inskip + idx * factorc + ch];
                out[1][saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
        } else {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][saveidx] += in[inskip + idx * factorc + ch];
                out[1][saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
            // save in two places -save in the extra bit
            out[0][saveidx + (gulpno * gulp * nchans)] = out[0][saveidx];
            out[1][saveidx + (gulpno * gulp * nchans)] = out[1][saveidx];
            out[2][saveidx + (gulpno * gulp * nchans)] = out[2][saveidx];
            out[3][saveidx + (gulpno * gulp * nchans)] = out[3][saveidx];
            }
        framet++;
    }
    // not a problem - earch thread in a warp uses the same branch
/*    if ((framet % totsize) < gulpno * gulp) {
        for (int ac = 0; ac < acc; ac++) {
            inskip = ac * 27 * 336;
            outskip = ac * 27 * 336 / factorc;
            for (int ch = 0; ch < factorc; ch++) {
                out[0][outskip + saveidx] += in[inskip + idx * factorc + ch];
                out[1][outskip + saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][outskip + saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][outskip + saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
        }
    } else {
        for (int ac = 0; ac < acc; ac++) {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][outskip + saveidx] += in[idx * factorc + ch];
                out[1][outskip + saveidx] += in[idx * factorc + ch + jumpin];
                out[2][outskip + saveidx] += in[idx * factorc + ch + 2 * jumpin];
                out[3][outskip + saveidx] += in[idx * factorc + ch + 3 * jumpin];
            }
            // save in two places - wrap wround to the start of the buffer
            out[0][outskip + saveidx - (gulpno * gulp * nchans)] = out[0][outskip + saveidx];
            out[1][outskip + saveidx - (gulpno * gulp * nchans)] = out[1][outskip + saveidx];
            out[2][outskip + saveidx - (gulpno * gulp * nchans)] = out[2][outskip + saveidx];
            out[3][outskop + saveidx - (gulpno * gulp * nchans)] = out[3][outskip + saveidx];
        }
    }
*/
}

__global__ void addchanscale(float* __restrict__ in, float** __restrict__ out, short nchans, size_t gulp, size_t totsize,  short gulpno, unsigned int jumpin, unsigned int factorc, unsigned int framet, unsigned int acc, float **means, float **rstdevs) {

    // the number of threads is equal to the number of output channels
    // each 'idx' is responsible for one output frequency channel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int extra = totsize - gulpno * gulp;
    float avgfactor = 1.0f / factorc;
    // thats the starting save position for the chunk of length acc time samples
    int saveidx;

    float tmp0, tmp1, tmp2, tmp3;   

    int inskip;

    for (int ac = 0; ac < acc; ac++) {
        // channels in increasing order
        // saveidx = (framet % (gulpno * gulp)) * nchans + idx;
        // channels in decreasing order
        saveidx = (framet % (gulpno * gulp)) * nchans + nchans - (idx + 1);
        inskip = ac * 27 * 336;

        out[0][saveidx] = (float)0.0;
        out[1][saveidx] = (float)0.0;
        out[2][saveidx] = (float)0.0;
        out[3][saveidx] = (float)0.0;

        // use scaling of the form
        // out = (in - mean) / stdev * 32 + 64;
        // rstdev = (1 / stdev) * 32 to reduce the number of operations
        if ((framet % (gulpno * gulp)) >= extra) {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][saveidx] += in[inskip + idx * factorc + ch];
                out[1][saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
            // scaling
            out[0][saveidx] = (out[0][saveidx] * avgfactor - means[0][idx]) * rstdevs[0][idx] + 64.0f;
            out[1][saveidx] = (out[1][saveidx] * avgfactor - means[1][idx]) * rstdevs[1][idx] + 64.0f;
            out[2][saveidx] = (out[2][saveidx] * avgfactor - means[2][idx]) * rstdevs[2][idx] + 64.0f;
            out[3][saveidx] = (out[3][saveidx] * avgfactor - means[3][idx]) * rstdevs[3][idx] + 64.0f;
        } else {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][saveidx] += in[inskip + idx * factorc + ch];
                out[1][saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
            // scaling
            out[0][saveidx] = (out[0][saveidx] * avgfactor - means[0][idx]) * rstdevs[0][idx] + 64.0f;
            out[1][saveidx] = (out[1][saveidx] * avgfactor - means[1][idx]) * rstdevs[1][idx] + 64.0f;
            out[2][saveidx] = (out[2][saveidx] * avgfactor - means[2][idx]) * rstdevs[2][idx] + 64.0f;
            out[3][saveidx] = (out[3][saveidx] * avgfactor - means[3][idx]) * rstdevs[3][idx] + 64.0f;
            tmp0 = rintf(fminf(fmaxf(0.0, out[0][saveidx]), 255.0));
            out[0][saveidx] = tmp0;
            //out[0][saveidx] = fminf(255, out[0][saveidx]);
            out[1][saveidx] = fmaxf(0.0, out[0][saveidx]);
            out[1][saveidx] = fminf(255, out[0][saveidx]);
            out[2][saveidx] = fmaxf(0.0, out[0][saveidx]);
            out[2][saveidx] = fminf(255, out[0][saveidx]);
            out[3][saveidx] = fmaxf(0.0, out[0][saveidx]);
            out[3][saveidx] = fminf(255, out[0][saveidx]);

            // save in two places -save in the extra bit
            out[0][saveidx + (gulpno * gulp * nchans)] = out[0][saveidx];
            out[1][saveidx + (gulpno * gulp * nchans)] = out[1][saveidx];
            out[2][saveidx + (gulpno * gulp * nchans)] = out[2][saveidx];
            out[3][saveidx + (gulpno * gulp * nchans)] = out[3][saveidx];
        }
        framet++;
    }

}
__global__ void powerscale(hipfftComplex *in, float *out, unsigned int jump)
{

    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx1 == 0) printf("In the power kernel\n");
    // offset introduced, jump to the B polarisation data - can cause some slowing down
    int idx2 = idx1 + jump;
    // these calculations assume polarisation is recorded in x,y base
    // i think the if statement is unnecessary as the number of threads for this
    // kernel 0s fftpoint * timeavg * nchans, which is exactly the size of the output array
    if (idx1 < jump) {      // half of the input data
        float power1 = (in[idx1].x * in[idx1].x + in[idx1].y * in[idx1].y) * fftfactor;
        float power2 = (in[idx2].x * in[idx2].x + in[idx2].y * in[idx2].y) * fftfactor;
        out[idx1] = (power1 + power2); // I; what was this doing here? / 2.0;
        //printf("Input numbers for %i and %i with jump %i: %f %f %f %f, with power %f\n", idx1, idx2, jump, in[idx1].x, in[idx1].y, in[idx2].x, in[idx2].y, out[idx1]);
        out[idx1 + jump] = (power1 - power2); // Q
        out[idx1 + 2 * jump] = 2 * fftfactor * (in[idx1].x * in[idx2].x + in[idx1].y * in[idx2].y); // U
        out[idx1 + 3 * jump] = 2 * fftfactor * (in[idx1].x * in[idx2].y - in[idx1].y * in[idx2].x); // V
    }
}

__global__ void powertime(hipfftComplex* __restrict__ in, float* __restrict__ out, unsigned int jump, unsigned int factort)
{
    // 1MHz channel ID
    int idx1 = blockIdx.x;
    // 'small' channel ID
    int idx2 = threadIdx.x;
    float power1;
    float power2;

    idx1 = idx1 * YSIZE * 2;
    int outidx = 27 * blockIdx.x + threadIdx.x;

    out[outidx] = (float)0.0;
    out[outidx + jump] = (float)0.0;
    out[outidx + 2 * jump] = (float)0.0;
    out[outidx + 3 * jump] = (float)0.0;

    for (int ii = 0; ii < factort; ii++) {
        idx2 = threadIdx.x + ii * 32;
	power1 = (in[idx1 + idx2].x * in[idx1 + idx2].x + in[idx1 + idx2].y * in[idx1 + idx2].y) * fftfactor;
        power2 = (in[idx1 + 128 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + 128 + idx2].y * in[idx1 + 128 + idx2].y) * fftfactor;
	out[outidx] += (power1 + power2);
        out[outidx + jump] += (power1 - power2);
        out[outidx + 2 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + idx2].y * in[idx1 + 128 + idx2].y));
        out[outidx + 3 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].y - in[idx1 + idx2].y * in[idx1 + 128 + idx2].x));

    }

   printf("%i, %i: %i\n", blockIdx.x, threadIdx.x, out[outidx]);
}

__global__ void powertime2(hipfftComplex* __restrict__ in, float* __restrict__ out, unsigned int jump, unsigned int factort, unsigned int acc) {

    int idx1, idx2;
    int outidx;
    int skip1, skip2;
    float power1, power2;
    float avgfactor= 1.0f / factort;

    for (int ac = 0; ac < acc; ac++) {
        skip1 = ac * 336 * 128 * 2;
        skip2 = ac * 336 * 27;
        for (int ii = 0; ii < 7; ii++) {
            outidx = skip2 + 7 * 27 * blockIdx.x + ii * 27 + threadIdx.x;
            out[outidx] = (float)0.0;
            out[outidx + jump] = (float)0.0;
            out[outidx + 2 * jump] = (float)0.0;
            out[outidx + 3 * jump] = (float)0.0;

            idx1 = skip1 + 256 * (blockIdx.x * 7 + ii);

            for (int jj = 0; jj < factort; jj++) {
                idx2 = threadIdx.x + jj * 32;
                power1 = (in[idx1 + idx2].x * in[idx1 + idx2].x + in[idx1 + idx2].y * in[idx1 + idx2].y) * fftfactor;
                power2 = (in[idx1 + 128 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + 128 + idx2].y * in[idx1 + 128 + idx2].y) * fftfactor;
        	out[outidx] += (power1 + power2) * avgfactor;
                out[outidx + jump] += (power1 - power2) * avgfactor;
                out[outidx + 2 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + idx2].y * in[idx1 + 128 + idx2].y)) * avgfactor;
                out[outidx + 3 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].y - in[idx1 + idx2].y * in[idx1 + 128 + idx2].x)) * avgfactor;
            }
        }
    }

//    printf("%i, %i: %i\n", blockIdx.x, threadIdx.x, out[outidx]);
}

// initialise the scale factors
// memset is slower than custom kernels and not safe for anything else than int
__global__ void initscalefactors(float **means, float **rstdevs, int stokes) {
    // the scaling is (in - mean) * rstdev + 64.0f
    // and I want to get the original in back in the first running
    // will therefore set the mean to 64.0f and rstdev to 1.0f

    // each thread responsible for one channel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int ii = 0; ii < stokes; ii++) {
        means[ii][idx] = 64.0f;
        rstdevs[ii][idx] = 1.0f;
    }
}

// filterbank data saved in the format t1c1,t1c2,t1c3,...
// need to transpose to t1c1,t2c1,t3c1,... for easy and efficient scaling kernel
__global__ void transpose(float* __restrict__ in, float* __restrict__ out, unsigned int nchans, unsigned int ntimes) {

    // very horrible implementation or matrix transpose
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int start = idx * ntimes;
    for (int tsamp = 0; tsamp < ntimes; tsamp++) {
        out[start + tsamp] = in[idx + tsamp * nchans];
    }
}

__global__ void scale_factors(float *in, float **means, float **rstdevs, unsigned int nchans, unsigned int ntimes, int param) {
    // calculates mean and standard deviation in every channel
    // assumes the data has been transposed

    // for now have one thread per frequency channel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float mean;
    float variance;

    float ntrec = 1.0f / (float)ntimes;
    float ntrec1 = 1.0f / (float)(ntimes - 1.0f);

    unsigned int start = idx * ntimes;
    mean = 0.0f;
    variance = 0.0;
    // two-pass solution for now
    for (int tsamp = 0; tsamp < ntimes; tsamp++) {
        mean += in[start + tsamp] * ntrec;
    }
    means[param][idx] = mean;

    for (int tsamp = 0; tsamp < ntimes; tsamp++) {
        variance += (in[start + tsamp] - mean) * (in[start + tsamp] - mean);
    }
    variance *= ntrec1;
    // reciprocal of standard deviation
    // multiplied by the desired standard deviation of the scaled data
    // reduces the number of operations that have to be done on the GPU
    rstdevs[param][idx] = rsqrtf(variance) * 32.0f;
    // to avoid inf when there is no data in the channel
    if (means[param][idx] == 0)
        rstdevs[param][idx] = 0;
}

__global__ void bandpass() {



}
