#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "errors.hpp"

#define INT_PER_LINE 2
#define NFPGAS 48
#define NCHAN_COARSE 336
#define NCHAN_FINE_IN 32
#define NCHAN_FINE_OUT 27
#define NACCUMULATE 128
#define NPOL 2
#define NSAMPS 4
#define NSAMPS_SUMMED 2
#define NCHAN_SUM 16
#define NSAMP_PER_PACKET 128
#define NCHAN_PER_PACKET 7

__global__ void UnpackKernel(int2 *__restrict__ in, hipfftComplex *__restrict__ out) {

    int skip = 0;

    __shared__ int2 accblock[896];

    int chan = 0;
    int time = 0;
    int line = 0;

    hipfftComplex cpol;
    int polint;

    int outskip = 0;

    for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
        // NOTE: This is skipping whole words as in will be cast to int2
        // skip = iacc * NCHAN_COARSE * NSAMP_PER_PACKET + blockIdx.x * NCHAN_PER_PACKET * NSAMP_PER_PACKET;

        skip = blockIdx.x * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NCHAN_PER_PACKET * NSAMP_PER_PACKET;

        for (int ichunk = 0; ichunk < 7; ++ichunk) {
            line = ichunk * blockDim.x + threadIdx.x;
            chan = line % 7;
            time = line / 7;
            accblock[chan * NSAMP_PER_PACKET + time] = in[skip + line];
        }

        __syncthreads();

        skip = NCHAN_COARSE * NSAMP_PER_PACKET * NACCUMULATE;

        outskip = blockIdx.x * 7 * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET;

        for (chan = 0; chan < NCHAN_PER_PACKET; ++chan) {
            polint = accblock[chan * NSAMP_PER_PACKET + threadIdx.x].y;
            cpol.x = static_cast<float>(static_cast<short>( ((polint & 0xff000000) >> 24) | ((polint & 0xff0000) >> 8) ));
            cpol.y = static_cast<float>(static_cast<short>( ((polint & 0xff00) >> 8) | ((polint & 0xff) << 8) ));
            out[outskip + threadIdx.x] = cpol;

            polint = accblock[chan * NSAMP_PER_PACKET + threadIdx.x].x;
            cpol.x = static_cast<float>(static_cast<short>( ((polint & 0xff000000) >> 24) | ((polint & 0xff0000) >> 8) ));
            cpol.y = static_cast<float>(static_cast<short>( ((polint & 0xff00) >> 8) | ((polint & 0xff) << 8) ));

            out[skip + outskip + threadIdx.x] = cpol;

            outskip += NSAMP_PER_PACKET * NACCUMULATE;
        }
    }
}

__global__ void DetectScrunchKernel(
                                            hipComplex* __restrict__ in, // PFTF <-- FFT output order
                                            float* __restrict__ out  // TF <-- Filterbank order
                                            )
{

  /**
   * This block is going to do 2 timesamples for all coarse channels.
   * The fine channels are dealt with by the lanes
   */

  // gridDim.x should be Nacc * 128 / (32 * nsamps_to_add) == 256

  __shared__ float freq_sum_buffer[NCHAN_FINE_OUT*NCHAN_COARSE]; // 9072 elements

  int warp_idx = threadIdx.x >> 0x5;
  int lane_idx = threadIdx.x & 0x1f;

  int pol_step = NCHAN_COARSE * NSAMPS * NCHAN_FINE_IN * NACCUMULATE;

  int nwarps_per_block = blockDim.x/warpSize;

  int offset_into_coarse_chan = blockIdx.x * NCHAN_FINE_IN * NSAMPS_SUMMED;

  //Drop first 3 fine channels and last two fine channels
  if ((lane_idx > 2) & (lane_idx < 30))
    {
      // This warp
      // first sample in inner dimension = (32 * 2 * blockIdx.x)

      // This warp will loop over coarse channels in steps of NWARPS per block
      // coarse_chan_idx (0,335)

      for (int coarse_chan_idx = warp_idx; coarse_chan_idx < NCHAN_COARSE; coarse_chan_idx += nwarps_per_block)
        {
          float real = 0.0f;
          float imag = 0.0f;
          int coarse_chan_jump = NACCUMULATE * NCHAN_FINE_IN * NSAMPS * coarse_chan_idx + offset_into_coarse_chan + lane_idx;
          for (int pol_idx=0; pol_idx<NPOL; ++pol_idx)
            {
              int offset = pol_step * pol_idx + coarse_chan_jump;
              for (int sample_idx=0; sample_idx<NSAMPS_SUMMED; ++sample_idx)
                {
                  //Get first channel
                  int read_idx = offset + sample_idx * NCHAN_FINE_IN;
                  hipComplex val = in[read_idx];
                  real += val.x * val.x;
                  imag += val.y * val.y;
                }
              // 3 is the leading dead lane count
              // sketchy
              freq_sum_buffer[coarse_chan_idx*NCHAN_FINE_OUT + lane_idx - 3] = real + imag;
            }
        }
    }

  __syncthreads();

  for (int start_chan=warp_idx*warpSize; start_chan < (NCHAN_FINE_OUT * NCHAN_COARSE - NCHAN_SUM); start_chan+=blockDim.x) // blockDim.x is multiple of 32
    {
      //float sum = freq_sum_buffer[start_chan];
      // 4 because we are summing 16 channels in a warp reduce
      for (int ii=0; ii<4; ++ii)
        {
          if (lane_idx < warpSize-(1<<ii)-1)
            {
              freq_sum_buffer[start_chan+lane_idx] += freq_sum_buffer[start_chan + lane_idx + (1<<ii)];
            }
        }
      if (lane_idx & 0x0f)
        {
          int out_chan = (start_chan + lane_idx)/16;
          out[NCHAN_FINE_OUT * NCHAN_COARSE / NCHAN_SUM * blockIdx.x + out_chan] = freq_sum_buffer[start_chan+lane_idx];
        }
    }
  return;
}


int main(int argc, char *argv[])
{
//     unsigned short polai;
//     unsigned short polaq;
//
//     unsigned short polbi;
//     unsigned short polbq;

    size_t toread = 8 * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NFPGAS * NACCUMULATE;
    unsigned char *codifarray = new unsigned char[toread];

    for (int ifpga = 0; ifpga < 48; ++ifpga) {
        for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {

            for (int isamp = 0; isamp < 128; ++isamp) {

                for (int ichan = 0; ichan < 7; ++ichan) {

                    // polai = ((ifpga << 10) | (isamp << 2) | 0x0);
                    // polaq = ((ifpga << 10) | (isamp << 2) | 0x2);
                    // polbi = ((ifpga << 10) | (isamp << 2) | 0x1);
                    // polbq = ((ifpga << 10) | (isamp << 2) | 0x3);

                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 0] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 1] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 2] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 3] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 4] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 5] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 6] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 7] = 0;

                }
            }
        }
    }

    unsigned char *devdata;
    cudaCheckError(hipMalloc((void**)&devdata, toread * sizeof(unsigned char)));
    cudaCheckError(hipMemcpy(devdata, codifarray, toread * sizeof(unsigned char), hipMemcpyHostToDevice));

    hipfftComplex *unpacked;
    cudaCheckError(hipMalloc((void**)&unpacked, toread / 8 * sizeof(hipfftComplex)));

    int sizes[] = {32};

    hipfftHandle fftplan;
    cufftCheckError(hipfftPlanMany(&fftplan, 1, sizes, NULL, 1, sizes[0], NULL, 1, sizes[0], HIPFFT_C2C, 336 * NACCUMULATE * 4));

    float *detected;
    cudaCheckError(hipMalloc((void**)&detected, toread / 8 * sizeof(float)));

    UnpackKernel<<<48, 128, 0>>>(reinterpret_cast<int2*>(devdata), unpacked);
    cufftCheckError(hipfftExecC2C(fftplan, unpacked, unpacked, HIPFFT_FORWARD));
    DetectScrunchKernel<<<2 * NACCUMULATE, 1024, 0>>>(unpacked, detected);

    cudaCheckError(hipDeviceSynchronize());

    return 0;

}
