#include "hip/hip_runtime.h"
#include <cstdlib>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "errors.hpp"

#define INT_PER_LINE 2
#define NFPGAS 48
#define NCHAN_COARSE 336
#define NCHAN_FINE_IN 32
#define NCHAN_FINE_OUT 27
#define NACCUMULATE 128
#define NPOL 2
#define NSAMPS 4
#define NSAMPS_SUMMED 2
#define NCHAN_SUM 16
#define NSAMP_PER_PACKET 128
#define NCHAN_PER_PACKET 7

__global__ void UnpackKernel(int2 *__restrict__ in, hipfftComplex *__restrict__ out) {

    int skip = 0;

    __shared__ int2 accblock[896];

    int chan = 0;
    int time = 0;
    int line = 0;

    hipfftComplex cpol;
    int polint;

    int outskip = 0;

    for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
        // NOTE: This is skipping whole words as in will be cast to int2
        // skip = iacc * NCHAN_COARSE * NSAMP_PER_PACKET + blockIdx.x * NCHAN_PER_PACKET * NSAMP_PER_PACKET;

        skip = blockIdx.x * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NCHAN_PER_PACKET * NSAMP_PER_PACKET;

        for (int ichunk = 0; ichunk < 7; ++ichunk) {
            line = ichunk * blockDim.x + threadIdx.x;
            chan = line % 7;
            time = line / 7;
            accblock[chan * NSAMP_PER_PACKET + time] = in[skip + line];
        }

        __syncthreads();

        skip = NCHAN_COARSE * NSAMP_PER_PACKET * NACCUMULATE;

        outskip = blockIdx.x * 7 * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET;

        for (chan = 0; chan < NCHAN_PER_PACKET; ++chan) {
            polint = accblock[chan * NSAMP_PER_PACKET + threadIdx.x].y;
            cpol.x = static_cast<float>(static_cast<short>( ((polint & 0xff000000) >> 24) | ((polint & 0xff0000) >> 8) ));
            cpol.y = static_cast<float>(static_cast<short>( ((polint & 0xff00) >> 8) | ((polint & 0xff) << 8) ));
            out[outskip + threadIdx.x] = cpol;

            polint = accblock[chan * NSAMP_PER_PACKET + threadIdx.x].x;
            cpol.x = static_cast<float>(static_cast<short>( ((polint & 0xff000000) >> 24) | ((polint & 0xff0000) >> 8) ));
            cpol.y = static_cast<float>(static_cast<short>( ((polint & 0xff00) >> 8) | ((polint & 0xff) << 8) ));

            out[skip + outskip + threadIdx.x] = cpol;

            outskip += NSAMP_PER_PACKET * NACCUMULATE;
        }
    }
}

__global__ void DetectScrunchKernel(
                                            hipComplex* __restrict__ in, // PFTF <-- FFT output order
                                            float* __restrict__ out  // TF <-- Filterbank order
                                            )
{
  /**
   * This block is going to do 2 timesamples for all coarse channels.
   * The fine channels are dealt with by the lanes
   */

  // gridDim.x should be Nacc * 128 / (32 * nsamps_to_add) == 256

  __shared__ float freq_sum_buffer[NCHAN_FINE_OUT*NCHAN_COARSE]; // 9072 elements

  int warp_idx = threadIdx.x >> 0x5;
  int lane_idx = threadIdx.x & 0x1f;
  int pol_offset = NCHAN_COARSE * NSAMPS * NCHAN_FINE_IN * NACCUMULATE;
  int coarse_chan_offet = NACCUMULATE * NCHAN_FINE_IN * NSAMPS;
  int block_offset = NCHAN_FINE_IN * NSAMPS_SUMMED * blockIdx.x;
  int nwarps_per_block = blockDim.x/warpSize;

  //Drop first 3 fine channels and last 2 fine channels
  if ((lane_idx > 2) & (lane_idx < 30))
    {
      // This warp 
      // first sample in inner dimension = (32 * 2 * blockIdx.x)
      // This warp will loop over coarse channels in steps of NWARPS per block coarse_chan_idx (0,335)

      for (int coarse_chan_idx = warp_idx; coarse_chan_idx < NCHAN_COARSE; coarse_chan_idx += nwarps_per_block)
        {
          float real = 0.0f;
          float imag = 0.0f;
          int base_offset = coarse_chan_offet * coarse_chan_idx + block_offset + lane_idx;

          for (int pol_idx=0; pol_idx<NPOL; ++pol_idx)
            {
              int offset = base_offset + pol_offset * pol_idx;
              for (int sample_idx=0; sample_idx<NSAMPS_SUMMED; ++sample_idx)
                {
                  //Get first channel
                  // IDX = NCHAN_COARSE * NSAMPS * NCHAN_FINE_IN * NACCUMULATE * pol_idx
                  // + NACCUMULATE * NCHAN_FINE_IN * NSAMPS * coarse_chan_idx
                  // + blockIdx.x * NCHAN_FINE_IN * NSAMPS_SUMMED
                  // + NCHAN_FINE_IN * sample_idx
                  // + lane_idx;
                  hipComplex val = in[offset + NCHAN_FINE_IN * sample_idx];
                  real += val.x * val.x;
                  imag += val.y * val.y;
                }
              // 3 is the leading dead lane count
              // sketchy
              freq_sum_buffer[coarse_chan_idx*NCHAN_FINE_OUT + lane_idx - 3] = real + imag;
            }
        }
    }

  __syncthreads();

  /** 
   * Here each warp will reduce 32 channels into 2 channels
   * The last warp will have a problem that there will only be 16 values to process
   * 
   */
  if (threadIdx.x <  (NCHAN_FINE_OUT * NCHAN_COARSE / NCHAN_SUM))
    {
      float sum = 0.0;
      for (int chan_idx = threadIdx.x * NCHAN_SUM; chan_idx < (threadIdx.x+1) * NCHAN_SUM; ++chan_idx)
        {
          sum += freq_sum_buffer[chan_idx];
        }
      out[NCHAN_FINE_OUT * NCHAN_COARSE / NCHAN_SUM * blockIdx.x + threadIdx.x] = sum;
    }
  return;
}

int main(int argc, char *argv[])
{
//     unsigned short polai;
//     unsigned short polaq;
//
//     unsigned short polbi;
//     unsigned short polbq;

    size_t toread = 8 * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NFPGAS * NACCUMULATE;
    unsigned char *codifarray = new unsigned char[toread];

    for (int ifpga = 0; ifpga < 48; ++ifpga) {
        for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {

            for (int isamp = 0; isamp < 128; ++isamp) {

                for (int ichan = 0; ichan < 7; ++ichan) {

			
                    // polai = ((ifpga << 10) | (isamp << 2) | 0x0);
                    // polaq = ((ifpga << 10) | (isamp << 2) | 0x2);
                    // polbi = ((ifpga << 10) | (isamp << 2) | 0x1);
                    // polbq = ((ifpga << 10) | (isamp << 2) | 0x3);

                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 0] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 1] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 2] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 3] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 4] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 5] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 6] = 0;
                    codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 7] = 0;

                    if((ifpga == 0) && (ichan == 0)) {
		        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 0] = 0;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 1] = 2;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 2] = 0;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 3] = 2;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 4] = 0;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 5] = 2;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 6] = 0;
                        codifarray[(ifpga * NCHAN_PER_PACKET * NSAMP_PER_PACKET * NACCUMULATE + iacc * NSAMP_PER_PACKET * NCHAN_PER_PACKET + isamp * NCHAN_PER_PACKET + ichan) * 8 + 7] = 2;
		    }
                }
            }
        }
    }

    unsigned char *devdata;
    cudaCheckError(hipMalloc((void**)&devdata, toread * sizeof(unsigned char)));
    cudaCheckError(hipMemcpy(devdata, codifarray, toread * sizeof(unsigned char), hipMemcpyHostToDevice));

    hipfftComplex *unpacked;
    cudaCheckError(hipMalloc((void**)&unpacked, toread / 8 * sizeof(hipfftComplex)));

    int sizes[] = {32};

    hipfftHandle fftplan;
    cufftCheckError(hipfftPlanMany(&fftplan, 1, sizes, NULL, 1, sizes[0], NULL, 1, sizes[0], HIPFFT_C2C, 336 * NACCUMULATE * 4));

    float *detected;
    cudaCheckError(hipMalloc((void**)&detected, NCHAN_COARSE * NCHAN_FINE_OUT / 16 * NACCUMULATE * 128 / 32 / NSAMPS_SUMMED * sizeof(float)));

    std::cout << "Running the kernels..." << std::endl;

    UnpackKernel<<<48, 128, 0>>>(reinterpret_cast<int2*>(devdata), unpacked);
    cufftCheckError(hipfftExecC2C(fftplan, unpacked, unpacked, HIPFFT_FORWARD));
    DetectScrunchKernel<<<2 * NACCUMULATE, 1024, 0>>>(unpacked, detected);

    cudaCheckError(hipDeviceSynchronize());

    std::cout << "Copying the data back..." << std::endl;

    float *dataarray = new float[NCHAN_COARSE * NCHAN_FINE_OUT / 16 * NACCUMULATE * 128 / 32 / NSAMPS_SUMMED];
    cudaCheckError(hipMemcpy(dataarray, detected, NCHAN_COARSE * NCHAN_FINE_OUT / 16 * NACCUMULATE * 128 / 32 / NSAMPS_SUMMED * sizeof(float), hipMemcpyDeviceToHost));

    std::ofstream outdata("detected.dat");

    if (!outdata) {
        std::cerr << "Could not create the output file!" << std::endl;
        exit(EXIT_FAILURE);
    }

    for (int isamp = 0; isamp < NACCUMULATE * 128 / 32 / NSAMPS_SUMMED; ++isamp) {
        for (int ichan = 0; ichan < NCHAN_COARSE * NCHAN_FINE_OUT / 16; ++ichan) {
	    outdata << dataarray[isamp * NCHAN_COARSE * NCHAN_FINE_OUT / 16 + ichan] << " ";
	}
        outdata << std::endl;
    }
    outdata.close();
    
    delete [] dataarray;
    cudaCheckError(hipFree(detected));
    cudaCheckError(hipFree(devdata));
    delete [] codifarray;

    return 0;

}
